#include <iostream>
#include <stdlib.h>

#ifdef KERNEL1
#include "kernel1.cuh"
#endif
#ifdef KERNEL2
#include "kernel2.cuh"
#endif
#ifdef KERNEL3
#include "kernel3.cuh"
#endif

#include "support.h"

int main(int argc, char *argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    std::cout << "\nSetting up the problem..." << std::endl;
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;

    if (argc == 1)
    {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    }
    else if (argc == 2)
    {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    }
    else if (argc == 4)
    {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    }
    else
    {
        std::cout << "    Invalid input parameters!" << std::endl;
        std::cout << "    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000" << std::endl;
        std::cout << "    Usage: ./sgemm-tiled <m>            # All matrices are m x m" << std::endl;
        std::cout << "    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n" << std::endl;
        exit(EXIT_FAILURE);
    }

    A_sz = matArow * matAcol;
    B_sz = matBrow * matBcol;
    C_sz = matArow * matBcol;

    CUDA_ERR_CHECK(hipHostMalloc((void **)&A_h, sizeof(float) * A_sz));
    for (unsigned int i = 0; i < A_sz; i++)
    {
        A_h[i] = (rand() % 100) / 100.00;
    }

    CUDA_ERR_CHECK(hipHostMalloc((void **)&B_h, sizeof(float) * B_sz));
    for (unsigned int i = 0; i < B_sz; i++)
    {
        B_h[i] = (rand() % 100) / 100.00;
    }

    CUDA_ERR_CHECK(hipHostMalloc((void **)&C_h, sizeof(float) * C_sz));

    stopTime(&timer);
    std::cout << elapsedTime(timer) << "s" << std::endl;
    std::cout << "    A: matArow x matAcol" << std::endl;
    std::cout << "    B: matBrow x matBcol" << std::endl;
    std::cout << "    C: matArow x matBcol" << std::endl;

    // Launch kernel using msplitm ---------------------------
    std::cout << "Launching kernel..." << std::endl;
    startTime(&timer);
    msplitm('N', 'N', matArow, matBcol, matBrow, 1.0f, A_h, matArow, B_h, matBrow, 0.0f, C_h, matBrow);

    CUDA_ERR_CHECK(hipDeviceSynchronize());
    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    // Verify correctness -----------------------------------------------------
    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);

    // Free memory ------------------------------------------------------------

    CUDA_ERR_CHECK(hipHostFree(A_h));
    CUDA_ERR_CHECK(hipHostFree(B_h));
    CUDA_ERR_CHECK(hipHostFree(C_h));

    return 0;
}
