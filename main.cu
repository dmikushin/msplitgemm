#include <stdio.h>
#include <stdlib.h>

#ifdef KERNEL1
#include "kernel1.cuh"
#endif
#ifdef KERNEL2
#include "kernel2.cuh"
#endif
#ifdef KERNEL3
#include "kernel3.cuh"
#endif
#ifdef KERNEL4
#include "kernel4.cuh"
#endif

#include "support.h"

int main(int argc, char *argv[])
{
    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem...");
    fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1)
    {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    }
    else if (argc == 2)
    {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    }
    else if (argc == 4)
    {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    }
    else
    {
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
               "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
               "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
               "\n");
        exit(0);
    }

    A_sz = matArow * matAcol;
    B_sz = matBrow * matBcol;
    C_sz = matArow * matBcol;

    hipHostMalloc((void **)&A_h, sizeof(float) * A_sz);
    for (unsigned int i = 0; i < A_sz; i++)
    {
        A_h[i] = (rand() % 100) / 100.00;
    }

    hipHostMalloc((void **)&B_h, sizeof(float) * B_sz);
    for (unsigned int i = 0; i < B_sz; i++)
    {
        B_h[i] = (rand() % 100) / 100.00;
    }

    hipHostMalloc((void **)&C_h, sizeof(float) * C_sz);

    stopTime(&timer);
    printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
           matBrow, matBcol, matArow, matBcol);

    // Launch kernel using msplitm ---------------------------
    printf("Launching kernel...");
    fflush(stdout);
    startTime(&timer);
    msplitm('N', 'N', matArow, matBcol, matBrow, 1.0f, A_h, matArow, B_h, matBrow, 0.0f, C_h, matBrow);

    cuda_ret = hipDeviceSynchronize();
    stopTime(&timer);
    printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);

    // Free memory ------------------------------------------------------------

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);

    return 0;
}
